
#include <hip/hip_runtime.h>
//#include <iostream>
//
//#include "cuda_utils.h"
//
//__global__ void saxpy(int n, float a, float *x, float *y)
//{
//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//
//    if (i < n)
//        y[i] = a * x[i] + y[i];
//}
//
//int main()
//{
//    int N = 1 << 20;
//    float *x, *y, *dx, *dy;
//
//    x = (float *) malloc(N * sizeof(float));
//    y = (float *) malloc(N * sizeof(float));
//
//    CHECK_CUDA_RESULT( cudaMalloc(&dx, N * sizeof(float)) );
//    CHECK_CUDA_RESULT( cudaMalloc(&dy, N * sizeof(float)) );
//
//    for (int i = 0; i < N; i++) {
//        x[i] = 1.0f;
//        y[i] = 2.0f;
//    }
//
//    CHECK_CUDA_RESULT( cudaMemcpy(dx, x, N * sizeof(float), cudaMemcpyHostToDevice) );
//    CHECK_CUDA_RESULT( cudaMemcpy(dy, y, N * sizeof(float), cudaMemcpyHostToDevice) );
//
//    saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, dx, dy);
//
//    CHECK_CUDA_RESULT( cudaMemcpy(y, dy, N * sizeof(float), cudaMemcpyDeviceToHost) );
//
//    float maxError = 0.0f;
//    for (int i = 0; i < N; i++)
//        maxError = max(maxError, abs(y[i] - 4.0f));
//
//    printf("Max error: %f\n", maxError);
//
//    CHECK_CUDA_RESULT( cudaFree(dx) );
//    CHECK_CUDA_RESULT( cudaFree(dy) );
//    free(x);
//    free(y);
//}
