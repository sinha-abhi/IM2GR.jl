#include "hip/hip_runtime.h"
#include <iostream>

#include "cuda_utils.h"
#include "utils.h"

#define MRI_FILE "../data/lgemri.txt"

__global__ void traverse() {

}

int main() {
	int *sz = new int[3];

	std::string fname(MRI_FILE);
	std::vector<std::vector<uint8_t>> mri_data(parse_file(sz, fname));

	int ne = sz[0] * sz[1] * sz[2]; // number of elements
	uint8_t *mri = (uint8_t *) malloc(ne * sizeof(uint8_t));
	flatten(mri, mri_data);

    return 0;
}
